#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------------*/
/*  FICHERO:       clasificacionSOM.cu									        */
/*  AUTOR:         Jorge Azorin								       			    */
/*													                            */
/*  RESUMEN												                        */
/*  ~~~~~~~												                        */
/* Ejercicio grupal para la clasificaci�n de patrones de entrada basada         */
/* en SOM utilizando GPUs                                                       */
/*----------------------------------------------------------------------------*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>


// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "clasificacionSOM.h"
#include <Windows.h>



#define ERROR_CHECK { hipError_t err; if ((err = hipGetLastError()) != hipSuccess) { printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__);}}

typedef LARGE_INTEGER timeStamp;
double getTime();

/*----------------------------------------------------------------------------*/
/*  FUNCION A PARALELIZAR  (versi�n secuencial-CPU)  				          */
/*	Implementa la clasificaci�n basada en SOM de un conjunto de patrones      */
/*  de entrada definidos en un fichero                                         */
/*----------------------------------------------------------------------------*/
int ClasificacionSOMCPU()
{
	float distancia;
	float distanciaMenor=MAXDIST;

	for (int np=0;np<Patrones.Cantidad;np++) // Recorrido de todos los patrones
	{
		distanciaMenor=MAXDIST;
		for (int y = 0; y<SOM.Alto; y++)			// Recorrido de todas las neuronas
		{
			for (int x = 0; x<SOM.Ancho; x++)
			{
				distancia=CalculaDistancia(y,x,np);     // CalculaDistancia entre neurona (y,x) y patr�n np
				for (int vy=-1;vy<2;vy++)               // Calculo en la vecindad
					for (int vx=-1;vx<2;vx++)
						if (vx != 0 && vy != 0)         // No comprobar con la misma neurona
						   distancia+=CalculaDistancia(y+vy,x+vx,np);
				if (distancia < distanciaMenor)
				{
					distanciaMenor=distancia;  // Neurona con menor distancia
					EtiquetaCPU[np]=SOM.Neurona[y][x].label;
				}

			}
		}

	}
	return OKCLAS;									// Simulaci�n CORRECTA
}

// ---------------------------------------------------------------
// ---------------------------------------------------------------
// FUNCION A IMPLEMENTAR POR EL GRUPO (paralelizaci�n de ClasificacionSOMCPU)
// ---------------------------------------------------------------
// ---------------------------------------------------------------


struct d_AUX
{
	int Ancho;
	int Alto;
	int Dimension;
	//float*** pesos; //las dos primeras dimensiones son la de la neurona, la tercera es la del peso
	//int** label;
	float* pesos;
	int* label;
};


struct d_sTPatrones
{
	int Cantidad;
	int Dimension;
	float * Pesos; //2d to 1d
};
typedef struct d_sTPatrones d_TPatrones;

__global__ void calculadorNeuronal(int* d_EtiquetaGPU, d_AUX d_SOM, d_TPatrones d_Patrones){

	float distancia = 0;
	float distanciaMenor = MAXDIST;
	int np = threadIdx.x + blockIdx.x * blockDim.x;

	int height = d_SOM.Alto;
	int width = d_SOM.Ancho;
	
	if (np < d_Patrones.Cantidad) {
		distanciaMenor = MAXDIST;
		for (int y = 0; y<d_SOM.Alto; y++) {			// Recorrido de todas las neuronas
			for (int x = 0; x<d_SOM.Ancho; x++) {
				//distancia = CalculaDistancia(y, x, np);     // CalculaDistancia entre neurona (y,x) y patr�n np
				distancia = 0;
				if (y >= 0 && y<d_SOM.Alto && x >= 0 && x<d_SOM.Ancho) {
					for (int i = 0; i < d_Patrones.Dimension; i++){
						//distancia += abs(d_SOM.Neurona[y][x].pesos[i] - d_Patrones.Pesos[np][i]);
						//distancia += abs(d_SOM.pesos[arrayIndex(y, x, i, d_SOM.Alto, d_SOM.Ancho)] - d_Patrones.Pesos[arrayIndex(np,i, d_Patrones.Cantidad)]); //CANTIDAD O DIMENSION?
						distancia += abs(d_SOM.pesos[y + x*height + i*height*width] - d_Patrones.Pesos[np + i* d_Patrones.Cantidad]);
						
					}
					distancia /= d_Patrones.Dimension;
				}
				
				for (int vy = -1; vy<2; vy++) {           // Calculo en la vecindad
					for (int vx = -1; vx < 2; vx++) {
						if (vx != 0 && vy != 0) {         // No comprobar con la misma neurona
							//distancia += CalculaDistancia(y + vy, x + vx, np);
							if (y >= 0 && y < d_SOM.Alto && x >= 0 && x < d_SOM.Ancho) {
								for (int i = 0; i < d_Patrones.Dimension; i++){
									distancia += abs(d_SOM.pesos[y + vy + (x + vx)*height + i*height*width] - d_Patrones.Pesos[np + i* d_Patrones.Cantidad]);
								}
								distancia /= d_Patrones.Dimension;
							}
						}
					}
				}
				if (distancia < distanciaMenor) {
					distanciaMenor = distancia;  // Neurona con menor distancia
					//d_EtiquetaGPU[np] = d_SOM.Neurona[y][x].label;
					d_EtiquetaGPU[np] = d_SOM.label[y + x* d_SOM.Alto];
					printf("Labelica wapa para %i: %i\n", np, d_EtiquetaGPU[np]);	
				}
			}
		}
	}
}

int getIndex(int x, int y, int z) {
	return x + y*SOM.Alto + z*SOM.Alto*SOM.Ancho;
}

int getIndex(int x, int y) {
	return x + y*SOM.Ancho;
}

__device__ int arrayIndex(int x, int y, int height) {
	return x + y*height;
}

__device__ int arrayIndex(int x, int y, int z, int height, int width) {
	return x + y*height + z*height*width;
}

int ClasificacionSOMGPU(){
	//TSOM SOM;
	//TPatrones Patrones;
	int * d_EtiquetaGPU = (int*)malloc(Patrones.Cantidad*sizeof(int));

	d_AUX aux;
	aux.Alto = SOM.Alto;
	aux.Ancho = SOM.Ancho;
	aux.Dimension = SOM.Dimension;

	int size = 0;
	for (int x = 0; x < SOM.Alto; x++){
		for (int y = 0; y < SOM.Ancho; y++){
			size += sizeof(SOM.Neurona[x][y].pesos);
		}
	}

	float* pesos = (float*)malloc(size);

	for (int i = 0; i < SOM.Ancho; i++){
		for (int j = 0; j < SOM.Alto; j++){
			for (int k = 0; k < sizeof(SOM.Neurona[i][j].pesos)/4; k++){
				pesos[getIndex(i, j, k)] = SOM.Neurona[i][j].pesos[k];
			}
		}
	}

	hipMalloc(&aux.pesos, size);
	hipMemcpy(aux.pesos, pesos, sizeof(pesos), hipMemcpyHostToDevice);

	int* labels = (int*)malloc(sizeof(int) * SOM.Alto * SOM.Ancho);

	for (int i = 0; i < SOM.Ancho; i++){
		for (int j = 0; j < SOM.Alto; j++){
			labels[getIndex(i, j)] = SOM.Neurona[i][j].label;
		}
	}

	hipMalloc(&aux.label, (sizeof(int) * SOM.Alto * SOM.Ancho));
	hipMemcpy(aux.label, labels, (sizeof(int) * SOM.Alto * SOM.Ancho), hipMemcpyHostToDevice);

	d_TPatrones d_Patrones;

	d_Patrones.Cantidad = Patrones.Cantidad;
	d_Patrones.Dimension = Patrones.Dimension;



	float* pesosP = (float*)malloc(Patrones.Cantidad * Patrones.Dimension * 4);

	for (int i = 0; i < Patrones.Cantidad; i++){
		for (int j = 0; j < Patrones.Dimension; j++){
			pesosP[getIndex(i, j)] = Patrones.Pesos[i][j];
		}
	}

	hipMalloc(&d_Patrones.Pesos, Patrones.Cantidad * Patrones.Dimension * 4);
	hipMemcpy(d_Patrones.Pesos, pesosP, size, hipMemcpyHostToDevice);

	hipMalloc(&d_EtiquetaGPU, Patrones.Cantidad*sizeof(int));

	dim3 block(16);
	dim3 grid((Patrones.Cantidad + (block.x - 1)) / block.x);
	calculadorNeuronal<<<grid,block>>>(EtiquetaGPU,aux,d_Patrones);

	hipMemcpy(EtiquetaGPU, d_EtiquetaGPU, Patrones.Cantidad*sizeof(int), hipMemcpyDeviceToHost);

	//printf("xddddd: %f", Patrones.Pesos[0][0]);
	return OKCLAS;

}

 // ---------------------------------------------------------------
 // ---------------------------------------------------------------
 // ---------------------------------------------------------------
 // ---------------------------------------------------------------
 // ---------------------------------------------------------------

 // Declaraciones adelantadas de funciones
 int LeerSOM(const char *fichero);
 int LeerPatrones(const char *fichero);
 int EscribirSOM(int ancho, int alto,int dimension, const char *fichero);
 int EscribirPatrones(int cantidad,int dimension, const char *fichero);



////////////////////////////////////////////////////////////////////////////////
//PROGRAMA PRINCIPAL
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char** argv)
{

  	double gpu_start_time, gpu_end_time;
	double cpu_start_time, cpu_end_time;

	//EscribirSOM(32,32,64,"peq.som");
	//EscribirPatrones(1024,64,"peq.pat");
	/* Numero de argumentos */
	if (argc != 3)
	{
		fprintf(stderr, "Numero de parametros incorecto\n");
		fprintf(stderr, "Uso: %s superficie pasossim\n", argv[0]);
		return;
	}

	/* Apertura de Fichero */
	printf("Clasificacion basada en SOM...\n");
	/* Mapa SOM */
	if (LeerSOM((char *)argv[1]) == ERRORCLASS)
	{
		fprintf(stderr, "Lectura de SOM incorrecta\n");
		return;
	}
	/* Patrones */
	if (LeerPatrones((char *)argv[2]) == ERRORCLASS)
	{
		fprintf(stderr, "Lectura de patrones incorrecta\n");
		return;
	}
	
	// Creaci�n etiquetas resultados para versiones CPU y GPU

	EtiquetaCPU = (int*)malloc(Patrones.Cantidad*sizeof(int));
	EtiquetaGPU = (int*)malloc(Patrones.Cantidad*sizeof(int));
	
	/* Algoritmo a paralelizar */
	cpu_start_time = getTime();
	if (ClasificacionSOMCPU() == ERRORCLASS)
	{
		fprintf(stderr, "Clasificaci�n CPU incorrecta\n");
		BorrarMapa();
		if (EtiquetaCPU != NULL) free(EtiquetaCPU);
		if (EtiquetaGPU != NULL) free(EtiquetaCPU);
		exit(1);
	}
	cpu_end_time = getTime();
	/* Algoritmo a implementar */
	gpu_start_time = getTime();
	if (ClasificacionSOMGPU() == ERRORCLASS)
	{
		fprintf(stderr, "Clasificaci�n GPU incorrecta\n");
		BorrarMapa();
		if (EtiquetaCPU != NULL) free(EtiquetaCPU);
		if (EtiquetaGPU != NULL) free(EtiquetaGPU);
		return;
	}
	gpu_end_time = getTime();
	// Comparaci�n de correcci�n
	int comprobar = OKCLAS;
	for (int i = 0; i<Patrones.Cantidad; i++)
	{
		if ((EtiquetaCPU[i] != EtiquetaGPU[i]))
		{
			comprobar = ERRORCLASS;
			fprintf(stderr, "Fallo en la clasificacion del patron %d, valor correcto %d\n", i, EtiquetaCPU[i]);
		}
	}
	// Impresion de resultados
	if (comprobar == OKCLAS)
	{
		printf("Clasificacion correcta!\n");

	}
	// Impresi�n de resultados
	printf("Tiempo ejecuci�n GPU : %fs\n", \
		gpu_end_time - gpu_start_time);
	printf("Tiempo de ejecuci�n en la CPU : %fs\n", \
		cpu_end_time - cpu_start_time);
	printf("Se ha conseguido un factor de aceleraci�n %fx utilizando CUDA\n", (cpu_end_time - cpu_start_time) / (gpu_end_time - gpu_start_time));
	// Limpieza de Neuronas
	BorrarMapa();
	BorrarPatrones();
	if (EtiquetaCPU != NULL) free(EtiquetaCPU);
	if (EtiquetaGPU != NULL) free(EtiquetaGPU);
	return;
}

int
main(int argc, char** argv)
{
	runTest(argc, argv);
	getchar();
}

/* Funciones auxiliares */
double getTime()
{
	timeStamp start;
	timeStamp dwFreq;
	QueryPerformanceFrequency(&dwFreq);
	QueryPerformanceCounter(&start);
	return double(start.QuadPart) / double(dwFreq.QuadPart);
}



/*----------------------------------------------------------------------------*/
/*	Funci�n:  LeerSOM(char *fichero)						              */
/*													                          */
/*	          Lee la estructura del SOM con formato .SOM   */
/*----------------------------------------------------------------------------*/
int LeerSOM(const char *fichero)
{
	int i, j, ndim, count;		/* Variables de bucle */
	int alto,ancho;		/* Variables de tama�o del mapa */
	FILE *fpin; 			/* Fichero */
	int nx,ny,lx,ly,label,dimension;
	float pesos;

	/* Apertura de Fichero */
	if ((fpin = fopen(fichero, "r")) == NULL) return ERRORCLASS;
	/* Lectura de cabecera */
	if (fscanf(fpin, "Alto: %d\n", &alto)<0) return ERRORCLASS;
	if (fscanf(fpin, "Ancho: %d\n", &ancho)<0) return ERRORCLASS;
	if (fscanf(fpin, "Dimension: %d\n", &dimension)<0) return ERRORCLASS;
	if (feof(fpin)) return ERRORCLASS;

	if (CrearMapa(alto, ancho, dimension) == ERRORCLASS) return ERRORCLASS;
	/* Lectura del SOM */
	count = 0;
	for (i = 0; i<ancho; i++)
	{
		for (j = 0; j<alto; j++)
		{
			if (!feof(fpin))
			{
				fscanf(fpin, "N%d,%d:", &nx, &ny);
				for (ndim = 0; ndim<dimension-1;ndim++)
				{
			        fscanf(fpin, " %f", &pesos);
				    SOM.Neurona[j][i].pesos[ndim] = pesos;
				}
				fscanf(fpin, " %f\n", &pesos);
				SOM.Neurona[j][i].pesos[ndim] = pesos;
				
				fscanf(fpin, "L%d,%d: %d\n", &lx, &ly, &label);
				SOM.Neurona[j][i].label=label;
			
				count++;
			}
			else break;
		}
	}
	fclose(fpin);
	if (count != ancho*alto) return ERRORCLASS;
	return OKCLAS;
}

/*----------------------------------------------------------------------------*/
/*	Funci�n:  LeerPatrones(char *fichero)						              */
/*													                          */
/*	          Lee los patrones de un fichero de entrada .pat   */
/*----------------------------------------------------------------------------*/
int LeerPatrones(const char *fichero)
{
	int i, ndim, count;		/* Variables de bucle */
	int cantidad,dimension;		/* Variables de tama�o de los patrones */
	FILE *fpin; 			/* Fichero */

	int np;
	float pesos;

	/* Apertura de Fichero */
	if ((fpin = fopen(fichero, "r")) == NULL) return ERRORCLASS;
	/* Lectura de cabecera */
	if (fscanf(fpin, "Numero: %d\n", &cantidad)<0) return ERRORCLASS;
	if (fscanf(fpin, "Dimension: %d\n", &dimension)<0) return ERRORCLASS;
	if (feof(fpin)) return ERRORCLASS;
	
	if (CrearPatrones(cantidad, dimension) == ERRORCLASS) return ERRORCLASS;
	/* Lectura de patrones */
	count = 0;
	for (i = 0; i<cantidad; i++)
	{
	  	if (!feof(fpin))
		{
			fscanf(fpin, "P%d:", &np);
				for (ndim = 0; ndim<dimension-1;ndim++)
				{
			        fscanf(fpin, " %f", &pesos);
					Patrones.Pesos[i][ndim] = pesos;
				}
				fscanf(fpin, " %f\n", &pesos);
				Patrones.Pesos[i][ndim] = pesos;
				
					
				count++;
			}
			else break;
	}
	
	fclose(fpin);
	if (count != cantidad) return ERRORCLASS;
	return OKCLAS;
}

int EscribirSOM(int alto, int ancho, int dimension,const char *fichero)
{
	int i, j, ndim, count;		/* Variables de bucle */
	
	FILE *fpin; 			/* Fichero */
	int label;
	float pesos;

	/* Apertura de Fichero */
	if ((fpin = fopen(fichero, "w")) == NULL) return ERRORCLASS;
	/* Lectura de cabecera */
	if (fprintf(fpin, "Alto: %d\n", alto)<0) return ERRORCLASS;
	if (fprintf(fpin, "Ancho: %d\n", ancho)<0) return ERRORCLASS;
	if (fprintf(fpin, "Dimension: %d\n", dimension)<0) return ERRORCLASS;
	if (feof(fpin)) return ERRORCLASS;

	count = 0;
	for (i = 0; i<alto; i++)
	{
		for (j = 0; j<ancho; j++)
		{
			if (!feof(fpin))
			{
				fprintf(fpin, "N%d,%d:", i+1, j+1);
				for (ndim = 0; ndim<dimension-1;ndim++)
				{
					pesos=(rand()%1000)/10.0;
			        fprintf(fpin, " %lf", pesos);
				}				
				pesos=(rand()%1000)/10;
				fprintf(fpin, " %lf\n", &pesos);
				
				label=(rand()%5);
				fprintf(fpin, "L%d,%d: %d\n", i+1, j+1, label);
				
				count++;
			}
			else break;
		}
	}
	fclose(fpin);
	if (count != ancho*alto) return ERRORCLASS;
	return OKCLAS;
}


/*----------------------------------------------------------------------------*/
/*	Funci�n:  LeerPatrones(char *fichero)						              */
/*													                          */
/*	          Lee los patrones de un fichero de entrada .pat   */
/*----------------------------------------------------------------------------*/
int EscribirPatrones(int cantidad, int dimension,const char *fichero)
{
	int i, ndim, count;		/* Variables de bucle */
	
	FILE *fpin; 			/* Fichero */

	int np;
	float pesos;

	/* Apertura de Fichero */
	if ((fpin = fopen(fichero, "w")) == NULL) return ERRORCLASS;
	/* Lectura de cabecera */
	if (fprintf(fpin, "Numero: %d\n", cantidad)<0) return ERRORCLASS;
	if (fprintf(fpin, "Dimension: %d\n", dimension)<0) return ERRORCLASS;
	if (feof(fpin)) return ERRORCLASS;
	
	/* Lectura de patrones */
	count = 0;
	for (i = 0; i<cantidad; i++)
	{
	  	if (!feof(fpin))
		{
			fprintf(fpin, "P%d:", i+1);
				for (ndim = 0; ndim<dimension-1;ndim++)
				{
					pesos=(rand()%1000)/10.0;
			        fprintf(fpin, " %lf", pesos);
				}
				pesos=(rand()%1000)/10;
				fprintf(fpin, " %lf\n", pesos);
					
					
				count++;
			}
			else break;
	}
	
	fclose(fpin);
	if (count != cantidad) return ERRORCLASS;
	return OKCLAS;
}

